#include <iostream>
#include <fstream>
#include <string>
#include <numeric>
#include <cmath>
#include <map>
#include <vector>
#include <stdlib.h>
#include <algorithm>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <unistd.h>

using namespace std;

// a point looks like (user, movie, date, rating)
#define POINT_SIZE 4 // the size of a single input point in the training data
#define STOPPING_CONDITION 0
#define MAX_EPOCHS 30 // the maximum number of epochs to run; 30 in the paper
#define MAX_NEIGHBOR_SIZE 300 // obtained from SVD++ paper
#define LAMBDA_7 0.0205 // tuned
#define LAMBDA_A 50 // from belkor paper 
#define LABMDA_pukt 0.01 // from bellkor paper
#define BETA 0.4 // from timeSVD++ paper
#define DECAY 0.9 // from belkor paper 
#define VAL_SET 4 // the point set being used for validation

// set learning rates and regularizers for all variables to be learned
// these values are taken from BellKor paper
#define learningRateBu 0.00267
#define learningRateBi 0.00048
#define learningRateBiBin 0.000115
#define learningRateAu 0.00000311
#define learningRateBut 0.00257
#define learningRateCu 0.00564
#define learningRateCut 0.00103
#define learningRateBifui 0.00236
#define regularizerBu 0.0255
#define regularizerBi 0.0255
#define regularizerBiBin 0.0929
#define regularizerAu 3.95
#define regularizerBut 0.00231
#define regularizerCu 0.0476
#define regularizerCut 0.0190
#define regularizerBifui 0.000000011
#define K 100.0

/*
* These are all the global variables used by the program
*/

// users and movies are one indexed
double num_users = 458293;
double num_movies = 17770;
double num_pts = 102416306;

// GPU version of number of points
double *d_num_pts;

// gamma_2 is the step size         
double GAMMA_2 = 0.008;
double *d_GAMMA_2; // GPU version of GAMMA 2

double GAMMA_pukt = 0.004;
double *d_GAMMA_pukt; // GPU version of GAMMA_pukt
// alpha step size; got default from timeSVD++ repo online
double GAMMA_A = 0.00001;
double *d_GAMMA_A; // GPU version of GAMMA_A

// though these are declared as single dimensional, I will use them as 2D arrays
// to facilitate this, I will store the sizes of the arrays as well
// we add one to the num_users because these arrays are 1-indexed
// these will be on the GPU because I will never need to access them on the host
double *user_values; // this is p in the SVD++ paper
int user_values_size = (int)((num_users + 1) * K);

double *movie_values; // this is q in the SVD++ paper
int movie_values_size = (int)((num_movies + 1) * K);

// the arrays to store the ratings and indices data
// note that ratings will be used as a 2D array as well
double *ratings;
int ratings_size = (int) (num_pts * POINT_SIZE);
double *d_ratings; // this is the GPU version of the ratings data

double *indices;
double *d_indices; // this is the GPU version of the indices data

// stores each user's neighborhoods
// functionally, this is a 2D array that stores for each user the id of the
// this is an int array because the double array is too large to work with
// movies they provided feedback for
int *neighborhoods;
int neighborhoods_size = (int) ((num_users + 1) * MAX_NEIGHBOR_SIZE);
int *d_neighborhoods; // this is the GPU version of the neighborhoods data

double *neighborhood_sizes;
double *d_neighborhood_sizes; // this is the GPU version of the neighborhood sizes data

// y is a 2D array that holds K features for each of the movies
// the plus one in the size derives from the fact that the movies are 1 indexed
double *y; // will be on the GPU
int y_size = (int) ((num_movies + 1) * K); 

// 1D array that stores the mean date of rating for each user
// is one indexed
double * t_u;
double *d_t_u; // the GPU version of this mean date of rating vector

// 2D array that stores the alpha value for each user FACTOR for SVD
// will be stored on the GPU
double * alphas;
int alphas_size = (int) ((num_users + 1) * K);


// store the day specific time dependent user SVD terms
// the first vector is indexed by user(1 indexed)
// the second layer is indexed by factor number (0 indexed)
// each element of the map maps a date to a bias term
// vector<vector<map<int, double> > > p_ukt;

/*
* Global variables for baseline prediction
*/

// the mean rating in point set 1
double mean_rating = 3.60861;
double *d_mean_rating;

// this value is taken from BellKor paper
int num_time_bins = 30;
int *d_num_time_bins;

// the maximum time value in the data
int max_time = 2243;
int *d_max_time;

// the size of a single bin
// add 1 to make sure we round up and not down
int binsize = max_time/num_time_bins + 1;
int *d_binsize;

// vector to store data from base set
vector<int> timeBins;

// stores the naive user bias term - will be on GPU 
double * user_biases;
// stores the naive movie biase term  - will be on GPU 
double * movie_biases;

// stores the alpha values that influence the user bias - will be on GPU 
double * bias_alphas;

// store the single day variables for user bias
// index in vector represents user (one indexed as usual)
// each int in the map is a date
// the double is the actual bias term 
vector<map<int, double> > Bu_t;

// store the bin bias terms  - will be on GPU 
double * Bi_bin;
int Bi_bin_size = (int)((num_movies + 1) * num_time_bins);

// store the stable c_u terms - will be on GPU 
double * c_u;

// store the time dependent c_ut terms
// set up very similarly to Bu_t
vector<map<int, double> > c_ut;


/*
* Initialize states for random number generation
*/
__global__ void init_states(unsigned int seed, hiprandState_t* states)
{
	/* we have to initialize the state */
  hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
              blockIdx.x, /* the sequence number should be different for each core (unless you want all
                             cores to get the same sequence of numbers for some reason - use thread id! */
              0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
              &states[blockIdx.x]);
}


/*
* Takes in an array of states and array of doubles and puts a random double into each
*/
__global__ void randoms(hiprandState_t* states, double* numbers)
{
	numbers[blockIdx.x] = 0.01 * (double)(hiprand(&states[blockIdx.x]) % 10)/sqrt((float)K);
}

/*
* Initializes all elements of the numbers array to 1.0
*/
__global__ void init_to_1(double *numbers)
{
	numbers[blockIdx.x] = 1.0;
}

/*
* Allocates memory for the device side
*/
inline void initialize_device()
{

	cout << "Initializing the device.\n";

	// allocate memory for all constants
	hipMalloc((void **)&d_GAMMA_2, sizeof(double));
	hipMalloc((void **)&d_GAMMA_pukt, sizeof(double));
	hipMalloc((void **)&d_GAMMA_A, sizeof(double));
	hipMalloc((void **)&d_num_pts, sizeof(double));

	hipMalloc((void **)&d_mean_rating, sizeof(double));
	hipMalloc((void **)&d_num_time_bins, sizeof(int));
	hipMalloc((void **)&d_max_time, sizeof(int));
	hipMalloc((void **)&d_binsize, sizeof(int));


	// initialize all constants on the GPU by copying the memory from host to 
	// device
	hipMemcpy(d_GAMMA_2, &GAMMA_2, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_GAMMA_pukt, &GAMMA_pukt, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_GAMMA_A, &GAMMA_A, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_num_pts, &num_pts, sizeof(double), hipMemcpyHostToDevice);

	hipMemcpy(d_mean_rating, &mean_rating, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_num_time_bins, &num_time_bins, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_max_time, &max_time, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_binsize, &binsize, sizeof(int), hipMemcpyHostToDevice);

	// now, allocate memory for the user factors, movie factors, user alpha factors, 
	// neighborhoods, neighborhood sizes, bias alpha factors, user bias factors, and movie bias
	// factors

	// these will later be intialized to stochastic values
	hipMalloc((void **)&user_values, user_values_size * sizeof(double));
	/* CUDA's random number library uses curandState_t to keep track of the seed value
     we will store a random state for every thread  */
	hiprandState_t* user_states;
	/* allocate space on the GPU for the random states */
	hipMalloc((void**) &user_states, user_values_size * sizeof(hiprandState_t));
	/* invoke the GPU to initialize all of the random states */
	init_states<<<user_values_size, 1>>>(time(0), user_states);
	/* Assign a random value to each user factor. */
	randoms<<<user_values_size, 1>>>(user_states, user_values);
	hipFree(user_states);


	hipMalloc((void **)&movie_values, movie_values_size * sizeof(double));
	/* CUDA's random number library uses curandState_t to keep track of the seed value
     we will store a random state for every thread  */
	hiprandState_t* movie_states;
	/* allocate space on the GPU for the random states */
	hipMalloc((void**) &movie_states, movie_values_size * sizeof(hiprandState_t));
	/* invoke the GPU to initialize all of the random states */
	init_states<<<movie_values_size, 1>>>(time(0), movie_states);
	/* Assign a random value to each movie factor. */
	randoms<<<movie_values_size, 1>>>(movie_states, movie_values);
	hipFree(movie_states);
	

	/* These 5 arrays are populated host side and then copied over */
	hipMalloc((void **)&d_ratings, ratings_size * sizeof(double));
	hipMalloc((void **)&d_indices, num_pts * sizeof(int));
	hipMalloc((void **)&d_neighborhoods, neighborhoods_size * sizeof(int));
	hipMalloc((void **)&d_neighborhood_sizes, (num_users + 1) * sizeof(double));
	hipMalloc((void **)&d_t_u, (num_users + 1) * sizeof(double));
	
	hipMalloc((void **)&y, y_size * sizeof(double));
	hipMemset((void **)&y, 0, y_size * sizeof(double));
	
	hipMalloc((void **)&alphas, alphas_size * sizeof(double));
	hipMemset((void **)&alphas, 0, alphas_size * sizeof(double));


	hipMalloc((void **)&user_biases, (num_users + 1) * sizeof(double));
	hipMemset((void **)&user_biases, 0, (num_users + 1) * sizeof(double));

	hipMalloc((void **)&movie_biases, (num_movies + 1) * sizeof(double));
	hipMemset((void **)&movie_biases, 0, (num_movies + 1) * sizeof(double));


	hipMalloc((void **)&bias_alphas, (num_users + 1) * sizeof(double));
	hipMemset((void **)&bias_alphas, 0, (num_users + 1) * sizeof(double));
	

	hipMalloc((void **)&c_u, (num_users + 1) * sizeof(double));
	init_to_1<<<(num_users + 1), 1>>>(c_u);

	hipMalloc((void **)&Bi_bin, (Bi_bin_size) * sizeof(double));
	hipMemset((void **)&Bi_bin, 0, (Bi_bin_size) * sizeof(double));
}

/*
* Allocates memory for the host side
*
*/
inline void initialize_host()
{
	cout << "Initializing the host.\n";

	// create  the arrays that store the ratings input data and the indexes
    ratings = new double[ratings_size];
    indices = new double[((int) num_pts)];

    neighborhoods = new int[neighborhoods_size];
    neighborhood_sizes = new double[(int)(num_users + 1)];

    t_u = new double[(int)num_users + 1];
}

/*
* Reads the input data into ratings, indices, neighborhoods, and time ratings. 
* Then, this function copies all of these into the GPU memory.
*/
inline void read_data()
{
    cout << "Reading in training data.\n";
    // read in ratings data
    fstream ratings_file("../ratings.bin", ios::in | ios::binary);
    ratings_file.read(reinterpret_cast<char *>(ratings), sizeof(double) * num_pts * POINT_SIZE);
    ratings_file.close();

    // read in index data
    fstream indices_file("../indices.bin", ios::in | ios::binary);
    indices_file.read(reinterpret_cast<char *>(indices), sizeof(double) * num_pts);
    indices_file.close();

    // read in neighborhod data
    fstream neighborhood_file("../neighborhoods_12345.bin", ios::in | ios::binary);
    neighborhood_file.read(reinterpret_cast<char *>(neighborhoods), sizeof(int) * (num_users + 1) * MAX_NEIGHBOR_SIZE);
    neighborhood_file.close();

    // read in the neighborhood size data
    fstream nsize_file ("../neighborhood_sizes_12345.bin", ios::in | ios::binary);
    nsize_file.read(reinterpret_cast<char *>(neighborhood_sizes), sizeof(double) * (num_users + 1));
    nsize_file.close();

    // read in the average rating time data
    fstream t_file("../average_time_rating.bin", ios::in | ios::binary);
    t_file.read(reinterpret_cast<char *>(t_u), sizeof(double) * (num_users + 1));
    t_file.close();

    // copy all of these host side arrays to the GPU
    hipMemcpy(d_ratings, &ratings, ratings_size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_indices, &indices, num_pts * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_neighborhoods, &neighborhoods, neighborhoods_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_neighborhood_sizes, &neighborhood_sizes, (num_users + 1) * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_t_u, &t_u, (num_users + 1) * sizeof(double), hipMemcpyHostToDevice);
}

/*
* Clears all used memory, on device and host.
*/
inline void clean_up()
{
	delete [] ratings;
	delete [] indices;
	delete [] neighborhoods;
	delete [] neighborhood_sizes;
	delete [] t_u;

	hipFree(d_GAMMA_2);
	hipFree(d_GAMMA_pukt);
	hipFree(d_GAMMA_A);
	hipFree(d_num_pts);

	hipFree(d_mean_rating);
	hipFree(d_num_time_bins);
	hipFree(d_max_time);
	hipFree(d_binsize);

	hipFree(d_ratings);
	hipFree(d_indices);
	hipFree(d_neighborhoods);
	hipFree(d_neighborhood_sizes);
	hipFree(d_t_u);

	hipFree(y);
	hipFree(alphas);
	hipFree(user_biases);
	hipFree(movie_biases);
	hipFree(bias_alphas);
	hipFree(c_u);
	hipFree(Bi_bin);	
	hipFree(user_values);
	hipFree(movie_values);	
}

int main()
{
	initialize_host();
	initialize_device();
	read_data();

	

	clean_up();
}

